/*
 * Copyright (c) 2020, Massimiliano Fasi and Mantas Mikaitis
 *
 * This program is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, version 2.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 *  You should have received a copy of the GNU General Public License along with
 *  this program. If not, see <http://www.gnu.org/licenses/>.
 */


#include <hip/hip_runtime.h>
#include <assert.h>
#include <unistd.h>
#include <cstdint>
#include <chrono>
#include <iostream>
#include <mma.h>
#include <iomanip>

using namespace nvcuda;

/*******************
 * Debug functions *
 *******************/
/* Print the elements of the m x n matrix A. The elements are assumed to be
   stored by columns if `bycols` is `true` and by rows if `bycols` is false. */
template <typename floattype>
void print_matrix (double *a,
                   size_t m, size_t n,
                   bool bycols) {
  int i, j;
  if (bycols) {
    for (i=0; i<m; i++) {
      for (j=0; j<n; j++)
        std::cout << a[j*n+i] << " ";
      std::cout << std::endl;
    }
    std::cout << std::endl;
  } else {
    for (i=0; i<m; i++ ) {
      for (j=0; j<n; j++)
        std::cout << a[i*m+j] << " ";
      std::cout  << std::endl;
    }
    std::cout << std::endl;
   }
}


/****************************************************
 * Memory management and wmma::mma_sync() interface *
 ****************************************************/

/* Set the entries of host arrays to zero. */
void host_reset(double *a, double *b, double *c) {
  memset(a, 0, 16*16*sizeof(double));
  memset(b, 0, 16*16*sizeof(double));
  memset(c, 0, 16*16*sizeof(double));
}

/* Compute C += A*B, where A, B, and C are 16x16x16 matrices.
   The matrix C is initialized to 0 when `init` is true. */
__global__ void wmma_ker(double *a, double *b, double *c, bool init) {

  // Declare fragments.
  wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> a_fragment;
  wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::col_major> b_fragment;
  wmma::fragment<wmma::accumulator, 8, 8, 4, double> c_fragment;

  // Load input matrices and initialize output (if required).
  wmma::load_matrix_sync(a_fragment, a, 16);
  wmma::load_matrix_sync(b_fragment, b, 16);
  if (init)
    wmma::fill_fragment(c_fragment, 0.0f);
  else
    wmma::load_matrix_sync(c_fragment, c, 16, wmma::mem_col_major);

  // Multiply
  wmma::mma_sync(c_fragment, a_fragment, b_fragment, c_fragment);

  // Store the output
  wmma::store_matrix_sync(c, c_fragment, 16, wmma::mem_col_major);
}

/* Copy data from host to device, perform the operation, and copy result back to
   host. */
void wmma_init_run (double *h_a, double *h_b, double *h_c,
                    double *d_a, double *d_b, double *d_c,
                    bool init) {

  // Copy input from host to device.
  hipMemcpy(d_a, h_a, 16*16*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, 16*16*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, 16*16*sizeof(double), hipMemcpyHostToDevice);

  // Perform matrix multiplication.
  wmma_ker<<<1,32>>>(d_a, d_b, d_c, init);

  // Copy result from device to host.
  hipMemcpy(h_c, d_c, 16*16*sizeof(float), hipMemcpyDeviceToHost);
}


/**********************
 * Printing functions *
 **********************/
void printheader(FILE *outfile, const char *string) {
  fprintf(outfile,
          "+--------------------------------------------------------------+\n");
  fprintf(outfile, "| %-60s |\n", string);
  fprintf(outfile,
          "+--------------------------------------------------------------+\n");
}
void printitem(FILE *outfile, const char *string) {
  fprintf(outfile, "  | %-49s", string);
}

void printpass(FILE *outfile, bool status) {
  if (status)
    fprintf(outfile, " [PASS] |\n");
  else
    fprintf(outfile, " [FAIL] |\n");
}
void printfooter(FILE *outfile) {
  fprintf(outfile,
          "  +----------------------------------------------------------+\n\n");
}


/***************
 * EXPERIMENTS *
 ***************/
int main(int argc, char** argv){

  // Declare pointers and allocate memory.
  double *h_a, *h_b, *h_c, *d_a, *d_b, *d_c,
    minsubnormal64 = ldexp(1., -1074), // smallest subnormal binary32
    belowone = nextafter(1., 0.) ,   // largest float smaller than 1.0
    gapbelowone = 1. - belowone,
    aboveone = nextafter(1., 2.),    // smallest float larger than 1.0
    belowtwo = 2. - ldexp(1., -52);   // largest float smaller than 2.0

  assert(belowone == 1. - ldexp(1., -53));
  assert(aboveone == 1. + ldexp(1., -52));

  h_a = new double[16*16];
  h_b = new double[16*16];
  h_c = new double[16*16];
 
  hipMalloc(&d_a, 16*16*sizeof(double));
  hipMalloc(&d_b, 16*16*sizeof(double));
  hipMalloc(&d_c, 16*16*sizeof(double));

  FILE *outfile = stdout;
  bool pass;

  printheader(outfile, "A. Support for subnormal numbers");// ;

   printitem(outfile, "*) Binary64 subnormals in input");
  host_reset(h_a, h_b, h_c);
  h_a[0] = minsubnormal64;
  h_b[0] = ldexp(1, 52);
  wmma_init_run(h_a, h_b, h_c, d_a, d_b, d_c, false);
  printpass(outfile, h_c[0]==ldexp(1., -1022));

  printitem(outfile, "*) Binary64 subnormals in output");
  host_reset(h_a, h_b, h_c);
  h_a[0] = ldexp(1., -1022);
  h_b[0] = ldexp(1., -1);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = h_c[0] == ldexp(1, -1023);
  h_a[0] = ldexp(1., -1022);
  h_b[0] = 1.0;
  h_c[0] = ldexp(-1., -1023);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = pass && (h_c[0] == ldexp(1, -1023));
  printpass(outfile, pass);

  printfooter(outfile);

  printheader(outfile, "B. Accuracy of the dot products ");// ;

  int i;
  printitem(outfile, "*) Products are accumulated in binary64 ");
  host_reset(h_a, h_b, h_c);
  pass = true;
  for (i=0; i<2; i++) {
    h_a[i] = 0.5;
    h_b[i] = ldexp(1, -53);
  }
  h_c[0] = 1.;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = pass && h_c[0] == 1;
  printpass(outfile, pass);

  printfooter(outfile);

  printheader(outfile, "C. Rounding modes in tensor core computations ");

  printitem(outfile, "*) Round-to-nearest for positive values ");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<2; i++) {
    h_a[i] = 1.0;
  }
  h_b[0] = 2.;
  h_b[1] = ldexp(1., -52) + ldexp(1., -53);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = h_c[0] == 2. + ldexp(1, -51);
  h_b[1] = ldexp(1., -53);
  h_c[0] = 0.0;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  printpass(outfile, pass && h_c[0] == 2.);

  printitem(outfile, "*) Round-to-nearest for negative values ");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<2; i++) {
    h_a[i] = 1.0;
  }
  h_b[0] = -2.;
  h_b[1] = -ldexp(1., -52) - ldexp(1., -53);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = h_c[0] == -2. - ldexp(1, -51);
  h_b[1] = -ldexp(1, -53);
  h_c[0] = 0.0;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  printpass(outfile, pass && h_c[0] == -2.0);

  printitem(outfile, "*) Round-to-nearest ties broken to even ");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = 1.0;
  }
  h_b[0] = 2.;
  h_b[1] = ldexp(1., -52);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = h_c[0] == 2.;
  h_b[0] = 2.+ldexp(1, -51);
  h_b[1] = ldexp(1, -52);
  h_c[0] = 0.0;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  printpass(outfile, pass && h_c[0] == 2.0 + ldexp(1, -50));

  printfooter(outfile);

  printheader(outfile, "D. Features of the accumulator");

  printitem(outfile, "1) Extra bits in the significand alignment");
  host_reset(h_a, h_b, h_c);
  h_a[0] = 1.0;
  h_b[0] = 1.0;
  h_c[0] = -belowone;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  assert(1 - belowone == ldexp(1., -53));
  assert(gapbelowone == ldexp(1., -53));
  printpass(outfile, h_c[0] == ldexp(1., -53));

  printitem(outfile, "2) Normalization in addition (after each add)");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<2; i++) {
    h_a[i] = 1.0;
    h_b[i] = ldexp(1, -53);
  }
  h_c[0] = 1. - ldexp(1., -53);
  assert(h_c[0] == belowone);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = h_c[0] == 1.;
  printpass(outfile, pass);

  printitem(outfile, "3) Normalization in subtraction");
  host_reset(h_a, h_b, h_c);
  h_a[0] = 1.0;
  h_a[1] = 1.0;
  h_b[0] = 1.0;
  h_b[1] = -ldexp(1., -53);
  h_c[0] = -1. + ldexp(1., -53);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  pass = pass && h_c[0] == 0.0;
  printpass(outfile, pass);

  printitem(outfile, "4) No extra bits for carry out");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<2; i++) {
    h_a[i] = 1.0;
    h_b[i] = 1.0;
  }
  pass = true;
  for (i=0; i<2; i++) {
    if (i>0)
      h_b[i-1] = 1.0;
    h_b[i] = ldexp(1., -52);
    h_c[0] = 1. + ldexp(1., -51) + ldexp(1., -52);
    wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
    pass = pass && (h_c[0] == 2. || (h_c[0] == (2. + ldexp(1, -50))));
  }

  printpass(outfile, pass);

  printitem(outfile, "5) Monotonicity of dot product");
  host_reset(h_a, h_b, h_c);
  h_a[0] = 1.0;
  h_b[0] = ldexp(1., -54);
  h_a[1] = 1.0;
  h_b[1] = ldexp(1., -53) + ldexp(1., -54);
  h_c[0] = 1. - ldexp(1., -53);
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  double partial = h_c[0];
  h_c[0] = 1.0;
  wmma_init_run (h_a, h_b, h_c, d_a, d_b, d_c, false);
  printpass(outfile, h_c[0] >= partial);

  printfooter(outfile);

  // Free dynamically allocated memory.
  //  free(h_a);
  //  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
